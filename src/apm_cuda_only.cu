/**
 *  * APPROXIMATE PATTERN MATCHING
 *   *
 *    * INF560 X2016
 *     */

#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/time.h>

#define APM_DEBUG 0

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

char *read_input_file(char * filename, int * size) {
    char * buf ;
    off_t fsize;
    int fd = 0 ;
    int n_bytes = 1 ;

    /* Open the text file */
    fd = open(filename, O_RDONLY);
    if (fd == -1) {
        fprintf(stderr, "Unable to open the text file <%s>\n", filename);
        return NULL;
    }

    /* Get the number of characters in the textfile */
    fsize = lseek(fd, 0, SEEK_END);
    lseek(fd, 0, SEEK_SET);

    /* TODO check return of lseek */

#if APM_DEBUG
    printf( "File length: %lld\n", fsize ) ;
#endif

    /* Allocate data to copy the target text */
    buf = (char *)malloc( fsize * sizeof ( char ) ) ;
    if (buf == NULL) {
        fprintf(stderr,
                "Unable to allocate %lld byte(s) for main array\n",
                fsize);
        return NULL;
    }

    n_bytes = read(fd, buf, fsize);
    if (n_bytes != fsize) {
        fprintf(stderr,
                "Unable to copy %lld byte(s) from text file (%d byte(s) copied)\n",
                fsize, n_bytes) ;
        return NULL ;
    }

#if APM_DEBUG
    printf( "Number of read bytes: %d\n", n_bytes ) ;
#endif

    *size = n_bytes ;

    close(fd);

    return buf;
}


#define MIN3(a, b, c) ((a) < (b) ? ((a) < (c) ? (a) : (c)) : ((b) < (c) ? (b) : (c)))

__global__
void cuda_levenshtein(char *s1, char *s2, int len, int * result, int n_max) {
    unsigned int x, y, lastdiag, olddiag;

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i<n_max){
        int column[100];
        for (y = 1; y <= len; y++) {
            column[y] = y;
        }

        for (x = 1; x <= len; x++) {

            column[0] = x;
            lastdiag = x-1 ;

            for (y = 1; y <= len; y++) {
                olddiag = column[y];
                column[y] = MIN3(
                        column[y] + 1,
                        column[y-1] + 1,
                        lastdiag + (s1[y-1+i] == s2[x-1] ? 0 : 1)
                );
                lastdiag = olddiag;
            }
        }
        result[i] = column[len];
    }
}




int levenshtein(char *s1, char *s2, int len, int * column) {
    unsigned int x, y, lastdiag, olddiag;

    for (y = 1; y <= len; y++) {
        column[y] = y;
    }

    for (x = 1; x <= len; x++) {

        column[0] = x;
        lastdiag = x-1 ;

        for (y = 1; y <= len; y++) {
            olddiag = column[y];
            column[y] = MIN3(
                    column[y] + 1,
                    column[y-1] + 1,
                    lastdiag + (s1[y-1] == s2[x-1] ? 0 : 1)
            );
            lastdiag = olddiag;
        }
    }
    return(column[len]);
}


int main(int argc, char ** argv) {
    char ** pattern ;
    int * scounts;
    int * displs;
    int step;
    char * filename ;
    int approx_factor = 0 ;
    int nb_patterns = 0 ;
    int i, j ;
    char * buf ;
    struct timeval t1, t2;
    double duration ;
    int * n_matches ;
    int n_bytes ;
    int rank;
    int size;

    int max_pat;

    int chunk_size;

    /* Check number of arguments */
    if (argc < 4) {

        printf("Usage: %s approximation_factor "
                       "dna_database pattern1 pattern2 ...\n",
               argv[0]);
        return 1;
    }
    n_bytes=0;
    /* Get the distance factor */
    approx_factor = atoi(argv[1]);

    /* Grab the filename containing the target text */
    filename = argv[2];

    /* Get the number of patterns that the user wants to search for */
    nb_patterns = argc - 3;

    /* Fill the pattern array */
    pattern = (char **)malloc( nb_patterns * sizeof( char * ) ) ;

    if (pattern == NULL) {
        fprintf(
                stderr,
                "Unable to allocate array of pattern of size %d\n",
                nb_patterns
        );
        return 1 ;
    }

    /* Grab the patterns */
    for (i = 0 ; i < nb_patterns ; i++) {
        int l ;
        l = strlen(argv[i+3]) ;

        if (l <= 0) {
            fprintf( stderr, "Error while parsing argument %d\n", i+3 ) ;
            return 1 ;
        }

        pattern[i] = (char *)malloc( (l+1) * sizeof( char ) ) ;

        if (pattern[i] == NULL) {
            fprintf( stderr, "Unable to allocate string of size %d\n", l ) ;
            return 1 ;
        }

        strncpy( pattern[i], argv[i+3], (l+1) ) ;
    }
    printf( "Approximate Pattern Mathing: "
                    "looking for %d pattern(s) in file %s w/ distance of %d\n",
            nb_patterns, filename, approx_factor );

    buf = read_input_file( filename, &n_bytes ) ;
    if ( buf == NULL ) {
        return 1 ;
    }

    /* Allocate the array of matches */
    n_matches = (int *)malloc( nb_patterns * sizeof( int ) ) ;
    if (n_matches == NULL) {
        fprintf(
                stderr,
                "Error: unable to allocate memory for %ldB\n",
                nb_patterns * sizeof( int )
        );
        return 1 ;
    }
    /*****
   *    * BEGIN MAIN LOOP
   *       ******/

    /* Timer start */
    gettimeofday(&t1, NULL);
    max_pat=0;
    for(i=0; i<nb_patterns; i++){
        max_pat=max_pat>strlen(pattern[i]) ? max_pat : strlen(pattern[i]);
    }
	printf("hello %d\n", n_bytes);
    for ( i = 0 ; i < nb_patterns ; i++ ) {
        int size_pattern = strlen(pattern[i]) ;
		printf("hello\n");
        n_matches[i] = 0 ;

        char * d_rcv_buf;
        char * d_pattern;
        int * d_result;
        int n_max = (n_bytes-size_pattern + 1);
        int result[n_max];
		hipMalloc((void **)&d_rcv_buf, n_bytes*sizeof(char));
        hipMalloc((void **)&d_pattern, size_pattern*sizeof(char));
        hipMalloc((void **)&d_result, n_max*sizeof(int));

        hipMemcpy(d_rcv_buf, buf, n_bytes*sizeof(char), hipMemcpyHostToDevice);
        hipMemcpy(d_pattern, pattern[i], size_pattern*sizeof(char), hipMemcpyHostToDevice);
        // à corriger
        cuda_levenshtein<<<1, 1024>>>(d_rcv_buf, d_pattern, size_pattern, d_result, n_max);
		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipDeviceSynchronize() );
        gpuErrchk(hipMemcpy(result, d_result, n_max*sizeof(int), hipMemcpyDeviceToHost));
		hipFree(d_rcv_buf);
		hipFree(d_pattern);
		hipFree(d_result);
		printf("hello %d \n", result[0]);
		int * column;
	    column=(int *) malloc((size_pattern+1)*sizeof(int));
        for ( j = 0 ; j < n_bytes ; j++ ) {
            int distance = 0 ;
            int s ;

#if APM_DEBUG
            if ( j % 100 == 0 )
            {
            printf( "Procesing byte %d (out of %d)\n", j, n_bytes ) ;
            }
#endif
	    
            s = size_pattern ;
            if ( n_bytes - j < size_pattern )
            {
                s = n_bytes - j ;
                distance = levenshtein( pattern[i], &buf[j], s, column )+size_pattern-s;
            }
            else{
                distance = result[j];
            }

            if ( distance <= approx_factor ) {
                n_matches[i]++ ;
            }
        }
        printf("%d matches from %d\n", n_matches[0], rank);
        free( column );
    }

    /* Timer stop */
    gettimeofday(&t2, NULL);

    duration = (t2.tv_sec -t1.tv_sec)+((t2.tv_usec-t1.tv_usec)/1e6);

    printf( "APM done in %lf s\n", duration ) ;

    /*****
    *    * END MAIN LOOP
    *       ******/

    for ( i = 0 ; i < nb_patterns ; i++ )
    {
        printf( "Number of matches for pattern <%s>: %d\n",
                pattern[i], n_matches[i] ) ;
    }
    return 0 ;
}